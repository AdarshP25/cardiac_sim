#include "hip/hip_runtime.h"
#include "reac_diff_sim.hpp"
#include "mech_sim.hpp"
#include <hip/hip_runtime.h>
#include <vector>
#include <fstream>
#include <iostream>
#include <cstring>
#include <thread>
#include <stdexcept>
#include <string>
#include <toml.hpp>

static void check(hipError_t e)
{
    if (e != hipSuccess)
        throw std::runtime_error(hipGetErrorString(e));
}


void initial_excitation(float *h_u, float *h_v, int nx, int ny)
{
    size_t N = static_cast<size_t>(nx) * ny;
    std::memset(h_u, 0, N * sizeof(float));
    std::memset(h_v, 0, N * sizeof(float));

    // Example excitation pattern
    int cx = nx / 2;
    int cy = ny / 2;
    for (int j = cy - 1; j < cy + 1; ++j) {
       for (int i = cx - 1; i < cx + 1; ++i) {
           if (i >= 0 && i < nx && j >=0 && j < ny) {
               size_t idx = static_cast<size_t>(j) * nx + i;
               h_u[idx] = 1.5f;
               h_v[idx] = 0.4f;
           }
       }
    }
}

int main()
{

    auto params = toml::parse("config.toml");
    // Simulation parameters
    const int nx = toml::find<int>(params, "simulation", "nx"), ny = toml::find<int>(params, "simulation", "ny"), nt = toml::find<int>(params, "simulation", "nt"); // Grid size, number of timesteps
    const float dt = toml::find<float>(params, "simulation", "dt");                    // Timestep size
    const bool mechanics_on = toml::find<bool>(params, "simulation", "mechanics_on"); // Whether to run mechanics simulation

    // Reaction-Diffusion parameters
    const float eps0 =  toml::find<float>(params, "voltage", "eps0"), a =  toml::find<float>(params, "voltage", "a"), k =  toml::find<float>(params, "voltage", "k");
    const float D =  toml::find<float>(params, "voltage", "D"), mu1 =  toml::find<float>(params, "voltage", "mu1"), mu2 =  toml::find<float>(params, "voltage", "mu2");
    const float k_T =  toml::find<float>(params, "voltage", "k_T");

    // Mechanics parameters
    const float ks_edge = toml::find<float>(params, "mechanics", "ks_edge");  // Stiffness for axial springs
    const float ks_boundary = toml::find<float>(params, "mechanics", "ks_boundary");
    const float fiber_angle = toml::find<float>(params, "mechanics", "fiber_angle"); // radians
    const float damping = toml::find<float>(params, "mechanics", "damping");
    const float ks_radial = 100.0f; // Stiffness for diagonal springs

    const float T0 = 50.0f;       // Maximum active tension (tune this - start comparable to ks_axial?)
    const float beta = 20.0f;    // Steepness of activation (tune this)
    const float ua = 0.15f;      // Activation threshold for u (tune this)
    const float active_force_scaling = 0.1f; // Scales tension to force (tune this)

    const int snapshot_interval = toml::find<int>(params, "simulation", "snapshot_interval"); // Interval for saving snapshots

    hipStream_t ioStream;
    check(hipStreamCreate(&ioStream));

    size_t N = static_cast<size_t>(nx) * ny;
    size_t bytes_rd = N * sizeof(float);
    size_t bytes_mech = N * sizeof(float2);

    // Pinned host memory buffers for asynchronous snapshot writing
    float *h_frame_rd[2]; 
    check(hipHostAlloc(&h_frame_rd[0], bytes_rd, hipHostMallocDefault));
    check(hipHostAlloc(&h_frame_rd[1], bytes_rd, hipHostMallocDefault));

    float2 *h_frame_mech[2];
    check(hipHostAlloc(&h_frame_mech[0], bytes_mech, hipHostMallocDefault));
    check(hipHostAlloc(&h_frame_mech[1], bytes_mech, hipHostMallocDefault));

    // RD Sim
    ReacDiffSim sim(nx, ny);
    float *h_u = new float[N];
    float *h_v = new float[N];
    float *h_Ta = new float[N];
    initial_excitation(h_u, h_v, nx, ny);
    
    check(hipMemcpy(sim.d_u, h_u, bytes_rd, hipMemcpyHostToDevice));
    check(hipMemcpy(sim.d_v, h_v, bytes_rd, hipMemcpyHostToDevice));
    delete[] h_u;
    delete[] h_v;

    float* fiber_angles = new float[N];
    for (int j = 0; j < ny; ++j) {
        for (int i = 0; i < nx; ++i) {
            fiber_angles[j * nx + i] = fiber_angle;
        }
    }
    MechSim mechSim(nx, ny, fiber_angles, damping);

    // Time iter
    int buf_idx = 0;
    for (int t = 1; t <= nt; ++t)
    {
        
        sim.step(D, dt, eps0, a, k, mu1, mu2, k_T);

        if (mechanics_on){
            mechSim.step(dt, ks_edge, ks_radial, ks_boundary, sim.d_Ta);
        }
        
        // Snapshot saving logic
        if (t % snapshot_interval == 0)
        {
            buf_idx = 1 - buf_idx; // Toggle buffer index

            check(hipMemcpyAsync(
                h_frame_rd[buf_idx],
                sim.d_u, // Source: device u field
                bytes_rd,
                hipMemcpyDeviceToHost,
                ioStream));

            check(hipMemcpyAsync(
                h_frame_mech[buf_idx],
                mechSim.d_pos_c,
                bytes_mech,
                hipMemcpyDeviceToHost,
                ioStream));

            
            std::thread([t, buf_idx, bytes_rd, bytes_mech, h_frame_rd, h_frame_mech, ioStream]() mutable // Capture mutable ioStream
                {
                    check(hipStreamSynchronize(ioStream)); // Wait for copies on ioStream to finish

                    
                    std::string filename_u = "data2/u_" + std::to_string(t) + ".bin";
                    std::ofstream outU(filename_u, std::ios::binary);
                    if(outU) {
                        outU.write(reinterpret_cast<char*>(h_frame_rd[buf_idx]), bytes_rd);
                    } else {
                         std::cerr << "Error opening file: " << filename_u << std::endl;
                    }

                    std::string filename_x = "data2/x_" + std::to_string(t) + ".bin";
                    std::ofstream outX(filename_x, std::ios::binary);
                     if(outX) {
                        outX.write(reinterpret_cast<char*>(h_frame_mech[buf_idx]), bytes_mech);
                    } else {
                        std::cerr << "Error opening file: " << filename_x << std::endl;
                    }
                }).detach();

            std::cout << "Time step: " << t << " (Snapshot scheduled)" << std::endl;
        }
    }

    // Cleanup
    check(hipDeviceSynchronize());
    check(hipStreamSynchronize(ioStream));


    hipHostFree(h_frame_rd[0]);
    hipHostFree(h_frame_rd[1]);
    hipHostFree(h_frame_mech[0]);
    hipHostFree(h_frame_mech[1]);

    check(hipStreamDestroy(ioStream));

    std::cout << "Simulation finished." << std::endl;
    return 0;
}